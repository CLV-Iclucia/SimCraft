#include "hip/hip_runtime.h"
//
// Created by creeper on 4/25/24.
//
#include <FluidSim/cuda/mgpcg.cuh>
#include <FluidSim/cuda/utils.h>
namespace fluid::cuda {
__constant__ double kTransferWights[4][4][4];
// 0 stands for solid, 1 stands for fluid
__global__ void PrecomputeDownSampleKernel(CudaSurfaceAccessor<uint8_t> surf,
                                           CudaSurfaceAccessor<uint8_t> surf_nxt, uint n) {
  get_and_restrict_tid_3d(x, y, z, n, n, n);
  uint8_t val_1 = surf.read<hipBoundaryModeZero>(x * 2, y * 2, z * 2);
  uint8_t val_2 = surf.read<hipBoundaryModeZero>(x * 2 + 1, y * 2, z * 2);
  uint8_t val_3 = surf.read<hipBoundaryModeZero>(x * 2, y * 2 + 1, z * 2);
  uint8_t val_4 = surf.read<hipBoundaryModeZero>(x * 2 + 1, y * 2 + 1, z * 2);
  uint8_t val_5 = surf.read<hipBoundaryModeZero>(x * 2, y * 2, z * 2 + 1);
  uint8_t val_6 = surf.read<hipBoundaryModeZero>(x * 2 + 1, y * 2, z * 2 + 1);
  uint8_t val_7 = surf.read<hipBoundaryModeZero>(x * 2, y * 2 + 1, z * 2 + 1);
  uint8_t val_8 = surf.read<hipBoundaryModeZero>(x * 2 + 1, y * 2 + 1, z * 2 + 1);
  surf_nxt.write(val_1 && val_2 && val_3 && val_4 && val_5 && val_6 && val_7 && val_8, x, y, z);
}
__global__ void RestrictKernel(CudaSurfaceAccessor<float> u,
                               CudaSurfaceAccessor<float> uc, uint n) {
  get_and_restrict_tid_3d(x, y, z, n, n, n);
  double sum = 0.0;
  for (int i = 0; i < 4; i++)
    for (int j = 0; j < 4; j++)
      for (int k = 0; k < 4; k++)
        sum += kTransferWights[i][j][k] * u.read<hipBoundaryModeZero>(x * 2 + i - 1, y * 2 + j - 1, z * 2 + k - 1);
  uc.write(static_cast<float>(sum), x, y, z);
}
__global__ void ProlongateKernel(CudaSurfaceAccessor<float> uc,
                                 CudaSurfaceAccessor<uint8_t> active,
                                 CudaSurfaceAccessor<float> u, uint n) {
  get_and_restrict_tid_3d(x, y, z, n, n, n);
  if (!active.read(x, y, z)) return;
  double sum = u.read(x, y, z);
  // use trilinear interpolation
  int x0 = (x - 1) / 2;
  int y0 = (y - 1) / 2;
  int z0 = (z - 1) / 2;
  auto active_000 = active.read<hipBoundaryModeZero>(x0, y0, z0);
  auto active_100 = active.read<hipBoundaryModeZero>(x0 + 1, y0, z0);
  auto active_010 = active.read<hipBoundaryModeZero>(x0, y0 + 1, z0);
  auto active_110 = active.read<hipBoundaryModeZero>(x0 + 1, y0 + 1, z0);
  auto active_001 = active.read<hipBoundaryModeZero>(x0, y0, z0 + 1);
  auto active_101 = active.read<hipBoundaryModeZero>(x0 + 1, y0, z0 + 1);
  auto active_011 = active.read<hipBoundaryModeZero>(x0, y0 + 1, z0 + 1);
  auto active_111 = active.read<hipBoundaryModeZero>(x0 + 1, y0 + 1, z0 + 1);
  auto tx = x * 0.5 - x0 - 0.25;
  auto ty = y * 0.5 - y0 - 0.25;
  auto tz = z * 0.5 - z0 - 0.25;
  auto w000 = (1.0 - tx) * (1.0 - ty) * (1.0 - tz);
  auto w100 = tx * (1.0 - ty) * (1.0 - tz);
  auto w010 = (1.0 - tx) * ty * (1.0 - tz);
  auto w110 = tx * ty * (1.0 - tz);
  auto w001 = (1.0 - tx) * (1.0 - ty) * tz;
  auto w101 = tx * (1.0 - ty) * tz;
  auto w011 = (1.0 - tx) * ty * tz;
  auto w111 = tx * ty * tz;
  sum += w000 * uc.read<hipBoundaryModeZero>(x0, y0, z0) * active_000;
  sum += w100 * uc.read<hipBoundaryModeZero>(x0 + 1, y0, z0) * active_100;
  sum += w010 * uc.read<hipBoundaryModeZero>(x0, y0 + 1, z0) * active_010;
  sum += w110 * uc.read<hipBoundaryModeZero>(x0 + 1, y0 + 1, z0) * active_110;
  sum += w001 * uc.read<hipBoundaryModeZero>(x0, y0, z0 + 1) * active_001;
  sum += w101 * uc.read<hipBoundaryModeZero>(x0 + 1, y0, z0 + 1) * active_101;
  sum += w011 * uc.read<hipBoundaryModeZero>(x0, y0 + 1, z0 + 1) * active_011;
  sum += w111 * uc.read<hipBoundaryModeZero>(x0 + 1, y0 + 1, z0 + 1) * active_111;
  u.write(static_cast<float>(sum), x, y, z);
}
__global__ void DampedJacobiKernel(CudaSurfaceAccessor<float> u,
                                   CudaSurfaceAccessor<float> u_buf,
                                   CudaSurfaceAccessor<uint8_t> active,
                                   CudaSurfaceAccessor<float> f, uint n, float alpha) {
  get_and_restrict_tid_3d(x, y, z, n, n, n);
  float u_old = u.read(x, y, z);
  uint8_t axp = active.read<hipBoundaryModeZero>(x - 1, y, z);
  uint8_t axn = active.read<hipBoundaryModeZero>(x + 1, y, z);
  uint8_t ayp = active.read<hipBoundaryModeZero>(x, y - 1, z);
  uint8_t ayn = active.read<hipBoundaryModeZero>(x, y + 1, z);
  uint8_t azp = active.read<hipBoundaryModeZero>(x, y, z - 1);
  uint8_t azn = active.read<hipBoundaryModeZero>(x, y, z + 1);
  auto cnt = static_cast<double>(axp + axn + ayp + ayn + azp + azn);
  float pxp = static_cast<float>(axp) * u.read<hipBoundaryModeClamp>(x - 1, y, z);
  float pxn = static_cast<float>(axn) * u.read<hipBoundaryModeClamp>(x + 1, y, z);
  float pyp = static_cast<float>(ayp) * u.read<hipBoundaryModeClamp>(x, y - 1, z);
  float pyn = static_cast<float>(ayn) * u.read<hipBoundaryModeClamp>(x, y + 1, z);
  float pzp = static_cast<float>(azp) * u.read<hipBoundaryModeClamp>(x, y, z - 1);
  float pzn = static_cast<float>(azn) * u.read<hipBoundaryModeClamp>(x, y, z + 1);
  float div = f.read(x, y, z);
  u_buf.write(
      (1.0 - kDampedJacobiOmega) * static_cast<double>(u_old) +
          kDampedJacobiOmega * static_cast<double>((pxp + pxn + pyp + pyn + pzp + pzn - div) / cnt),
      x, y, z);
}

void bottomSolve() {

}

void prepareWeights() {
  double weights[4][4][4];
  for (auto &wi : weights)
    for (auto &wij : wi)
      for (double &wijk : wij)
        wijk = 1.0;
  for (int i = 0; i < 4; i++)
    for (auto &wij : weights[i])
      for (auto &wijk : wij) {
        if (i == 0 || i == 3) wijk *= 0.125;
        else if (i == 1 || i == 2) wijk *= 0.375;
      }
  for (auto &wi : weights)
    for (int j = 0; j < 4; j++)
      for (auto &wijk : wi[j]) {
        if (j == 0 || j == 3) wijk *= 0.125;
        else if (j == 1 || j == 2) wijk *= 0.375;
      }
  for (auto &wi : weights)
    for (auto &wij : wi)
      for (int k = 0; k < 4; k++) {
        if (k == 0 || k == 3) wij[k] *= 0.125;
        else if (k == 1 || k == 2) wij[k] *= 0.375;
      }
  hipMemcpyToSymbol(HIP_SYMBOL(kTransferWights), weights, sizeof(weights));
}
}