#include "hip/hip_runtime.h"
//
// Created by creeper on 4/25/24.
//
#include <FluidSim/cuda/mgpcg.cuh>
#include <FluidSim/cuda/utils.h>
namespace fluid::cuda {
__constant__ double kTransferWights[4][4][4];
// 0 stands for solid, 1 stands for fluid
__global__ void PrecomputeDownSampleKernel(CudaSurfaceAccessor<uint8_t> surf,
                                           CudaSurfaceAccessor<uint8_t> surf_nxt, uint n) {
  get_and_restrict_tid_3d(x, y, z, n, n, n);
  uint8_t val_1 = surf.read<hipBoundaryModeZero>(x * 2, y * 2, z * 2);
  uint8_t val_2 = surf.read<hipBoundaryModeZero>(x * 2 + 1, y * 2, z * 2);
  uint8_t val_3 = surf.read<hipBoundaryModeZero>(x * 2, y * 2 + 1, z * 2);
  uint8_t val_4 = surf.read<hipBoundaryModeZero>(x * 2 + 1, y * 2 + 1, z * 2);
  uint8_t val_5 = surf.read<hipBoundaryModeZero>(x * 2, y * 2, z * 2 + 1);
  uint8_t val_6 = surf.read<hipBoundaryModeZero>(x * 2 + 1, y * 2, z * 2 + 1);
  uint8_t val_7 = surf.read<hipBoundaryModeZero>(x * 2, y * 2 + 1, z * 2 + 1);
  uint8_t val_8 = surf.read<hipBoundaryModeZero>(x * 2 + 1, y * 2 + 1, z * 2 + 1);
  surf_nxt.write(val_1 && val_2 && val_3 && val_4 && val_5 && val_6 && val_7 && val_8, x, y, z);
}
__global__ void RestrictKernel(CudaSurfaceAccessor<float> u,
                               CudaSurfaceAccessor<float> uc, uint n) {
  get_and_restrict_tid_3d(x, y, z, n, n, n);
  double sum = 0.0;
  for (int i = 0; i < 4; i++)
    for (int j = 0; j < 4; j++)
      for (int k = 0; k < 4; k++)
        sum += kTransferWights[i][j][k] * u.read<hipBoundaryModeZero>(x * 2 + i - 1, y * 2 + j - 1, z * 2 + k - 1);
  uc.write(static_cast<float>(sum), x, y, z);
}
__global__ void ProlongateKernel(CudaSurfaceAccessor<float> uc,
                                 CudaSurfaceAccessor<uint8_t> active,
                                 CudaSurfaceAccessor<float> u, uint n) {
  get_and_restrict_tid_3d(x, y, z, n, n, n);
  if (!active.read(x, y, z)) return;
  double sum = u.read(x, y, z);
  // use trilinear interpolation

}
__global__ void DampedJacobiKernel(CudaSurfaceAccessor<float> u,
                                   CudaSurfaceAccessor<uint8_t> active,
                                   CudaSurfaceAccessor<float> f, uint n, float alpha) {
  get_and_restrict_tid_3d(x, y, z, n, n, n);
  double delta = 0.0;

}

void bottomSolve

void prepareWeights() {
  double weights[4][4][4];
  for (auto &wi : weights)
    for (auto &wij : wi)
      for (double &wijk : wij)
        wijk = 1.0;
  for (int i = 0; i < 4; i++)
    for (auto &wij : weights[i])
      for (auto &wijk : wij) {
        if (i == 0 || i == 3) wijk *= 0.125;
        else if (i == 1 || i == 2) wijk *= 0.375;
      }
  for (auto &wi : weights)
    for (int j = 0; j < 4; j++)
      for (auto &wijk : wi[j]) {
        if (j == 0 || j == 3) wijk *= 0.125;
        else if (j == 1 || j == 2) wijk *= 0.375;
      }
  for (auto &wi : weights)
    for (auto &wij : wi)
      for (int k = 0; k < 4; k++) {
        if (k == 0 || k == 3) wij[k] *= 0.125;
        else if (k == 1 || k == 2) wij[k] *= 0.375;
      }
  hipMemcpyToSymbol(HIP_SYMBOL(kTransferWights), weights, sizeof(weights));
}
}