#include "hip/hip_runtime.h"
//
// Created by creeper on 4/25/24.
//
#include <FluidSim/cuda/mgpcg.cuh>
#include <FluidSim/cuda/utils.h>
#include <Core/debug.h>
namespace fluid::cuda {
__constant__ double kTransferWights[4][4][4];
// 0 stands for solid, 1 stands for fluid
__global__ void PrecomputeDownSampleKernel(CudaSurfaceAccessor<uint8_t> surf,
                                           CudaSurfaceAccessor<uint8_t> surf_nxt, uint n) {
  get_and_restrict_tid_3d(x, y, z, n, n, n);
  uint8_t val_1 = surf.read<hipBoundaryModeZero>(x * 2, y * 2, z * 2);
  uint8_t val_2 = surf.read<hipBoundaryModeZero>(x * 2 + 1, y * 2, z * 2);
  uint8_t val_3 = surf.read<hipBoundaryModeZero>(x * 2, y * 2 + 1, z * 2);
  uint8_t val_4 = surf.read<hipBoundaryModeZero>(x * 2 + 1, y * 2 + 1, z * 2);
  uint8_t val_5 = surf.read<hipBoundaryModeZero>(x * 2, y * 2, z * 2 + 1);
  uint8_t val_6 = surf.read<hipBoundaryModeZero>(x * 2 + 1, y * 2, z * 2 + 1);
  uint8_t val_7 = surf.read<hipBoundaryModeZero>(x * 2, y * 2 + 1, z * 2 + 1);
  uint8_t val_8 = surf.read<hipBoundaryModeZero>(x * 2 + 1, y * 2 + 1, z * 2 + 1);
  surf_nxt.write(val_1 && val_2 && val_3 && val_4 && val_5 && val_6 && val_7 && val_8, x, y, z);
}
static __global__ void ComputeResidualKernel(CudaSurfaceAccessor<float> u,
                                             CudaSurfaceAccessor<float> b,
                                             CudaSurfaceAccessor<float> r,
                                             CudaSurfaceAccessor<uint8_t> active, uint n) {
  get_and_restrict_tid_3d(x, y, z, n, n, n);
  if (!active.read(x, y, z)) return;
  float u_old = u.read(x, y, z);
  uint8_t axp = active.read<hipBoundaryModeZero>(x - 1, y, z);
  uint8_t axn = active.read<hipBoundaryModeZero>(x + 1, y, z);
  uint8_t ayp = active.read<hipBoundaryModeZero>(x, y - 1, z);
  uint8_t ayn = active.read<hipBoundaryModeZero>(x, y + 1, z);
  uint8_t azp = active.read<hipBoundaryModeZero>(x, y, z - 1);
  uint8_t azn = active.read<hipBoundaryModeZero>(x, y, z + 1);
  auto cnt = static_cast<double>(axp + axn + ayp + ayn + azp + azn);
  float pxp = static_cast<float>(axp) * u.read<hipBoundaryModeClamp>(x - 1, y, z);
  float pxn = static_cast<float>(axn) * u.read<hipBoundaryModeClamp>(x + 1, y, z);
  float pyp = static_cast<float>(ayp) * u.read<hipBoundaryModeClamp>(x, y - 1, z);
  float pyn = static_cast<float>(ayn) * u.read<hipBoundaryModeClamp>(x, y + 1, z);
  float pzp = static_cast<float>(azp) * u.read<hipBoundaryModeClamp>(x, y, z - 1);
  float pzn = static_cast<float>(azn) * u.read<hipBoundaryModeClamp>(x, y, z + 1);
  r.write(b.read(x, y, z) - (pxp + pxn + pyp + pyn + pzp + pzn), x, y, z);
}
__global__ void RestrictKernel(CudaSurfaceAccessor<float> u,
                               CudaSurfaceAccessor<float> uc, uint n) {
  get_and_restrict_tid_3d(x, y, z, n, n, n);
  double sum = 0.0;
  for (int i = 0; i < 4; i++)
    for (int j = 0; j < 4; j++)
      for (int k = 0; k < 4; k++)
        sum += kTransferWights[i][j][k] * u.read<hipBoundaryModeZero>(x * 2 + i - 1, y * 2 + j - 1, z * 2 + k - 1);
  uc.write(static_cast<float>(sum), x, y, z);
}
__global__ void ProlongateKernel(CudaSurfaceAccessor<float> uc,
                                 CudaSurfaceAccessor<uint8_t> active,
                                 CudaSurfaceAccessor<float> u, uint n) {
  get_and_restrict_tid_3d(x, y, z, n, n, n);
  if (!active.read(x, y, z)) return;
  double sum = u.read(x, y, z);
  // use trilinear interpolation
  int x0 = (x - 1) / 2;
  int y0 = (y - 1) / 2;
  int z0 = (z - 1) / 2;
  auto active_000 = active.read<hipBoundaryModeZero>(x0, y0, z0);
  auto active_100 = active.read<hipBoundaryModeZero>(x0 + 1, y0, z0);
  auto active_010 = active.read<hipBoundaryModeZero>(x0, y0 + 1, z0);
  auto active_110 = active.read<hipBoundaryModeZero>(x0 + 1, y0 + 1, z0);
  auto active_001 = active.read<hipBoundaryModeZero>(x0, y0, z0 + 1);
  auto active_101 = active.read<hipBoundaryModeZero>(x0 + 1, y0, z0 + 1);
  auto active_011 = active.read<hipBoundaryModeZero>(x0, y0 + 1, z0 + 1);
  auto active_111 = active.read<hipBoundaryModeZero>(x0 + 1, y0 + 1, z0 + 1);
  auto tx = x * 0.5 - x0 - 0.25;
  auto ty = y * 0.5 - y0 - 0.25;
  auto tz = z * 0.5 - z0 - 0.25;
  auto w000 = (1.0 - tx) * (1.0 - ty) * (1.0 - tz);
  auto w100 = tx * (1.0 - ty) * (1.0 - tz);
  auto w010 = (1.0 - tx) * ty * (1.0 - tz);
  auto w110 = tx * ty * (1.0 - tz);
  auto w001 = (1.0 - tx) * (1.0 - ty) * tz;
  auto w101 = tx * (1.0 - ty) * tz;
  auto w011 = (1.0 - tx) * ty * tz;
  auto w111 = tx * ty * tz;
  sum += w000 * uc.read<hipBoundaryModeZero>(x0, y0, z0) * active_000;
  sum += w100 * uc.read<hipBoundaryModeZero>(x0 + 1, y0, z0) * active_100;
  sum += w010 * uc.read<hipBoundaryModeZero>(x0, y0 + 1, z0) * active_010;
  sum += w110 * uc.read<hipBoundaryModeZero>(x0 + 1, y0 + 1, z0) * active_110;
  sum += w001 * uc.read<hipBoundaryModeZero>(x0, y0, z0 + 1) * active_001;
  sum += w101 * uc.read<hipBoundaryModeZero>(x0 + 1, y0, z0 + 1) * active_101;
  sum += w011 * uc.read<hipBoundaryModeZero>(x0, y0 + 1, z0 + 1) * active_011;
  sum += w111 * uc.read<hipBoundaryModeZero>(x0 + 1, y0 + 1, z0 + 1) * active_111;
  u.write(static_cast<float>(sum), x, y, z);
}
__global__ void DampedJacobiKernel(CudaSurfaceAccessor<float> u,
                                   CudaSurfaceAccessor<float> u_buf,
                                   CudaSurfaceAccessor<uint8_t> active,
                                   CudaSurfaceAccessor<float> f, uint n) {
  get_and_restrict_tid_3d(x, y, z, n, n, n);
  float u_old = u.read(x, y, z);
  uint8_t axp = active.read<hipBoundaryModeZero>(x - 1, y, z);
  uint8_t axn = active.read<hipBoundaryModeZero>(x + 1, y, z);
  uint8_t ayp = active.read<hipBoundaryModeZero>(x, y - 1, z);
  uint8_t ayn = active.read<hipBoundaryModeZero>(x, y + 1, z);
  uint8_t azp = active.read<hipBoundaryModeZero>(x, y, z - 1);
  uint8_t azn = active.read<hipBoundaryModeZero>(x, y, z + 1);
  auto cnt = static_cast<double>(axp + axn + ayp + ayn + azp + azn);
  float pxp = static_cast<float>(axp) * u.read<hipBoundaryModeClamp>(x - 1, y, z);
  float pxn = static_cast<float>(axn) * u.read<hipBoundaryModeClamp>(x + 1, y, z);
  float pyp = static_cast<float>(ayp) * u.read<hipBoundaryModeClamp>(x, y - 1, z);
  float pyn = static_cast<float>(ayn) * u.read<hipBoundaryModeClamp>(x, y + 1, z);
  float pzp = static_cast<float>(azp) * u.read<hipBoundaryModeClamp>(x, y, z - 1);
  float pzn = static_cast<float>(azn) * u.read<hipBoundaryModeClamp>(x, y, z + 1);
  float div = f.read(x, y, z);
  u_buf.write(
      (1.0 - kDampedJacobiOmega) * static_cast<double>(u_old) +
          kDampedJacobiOmega * static_cast<double>((pxp + pxn + pyp + pyn + pzp + pzn - div) / cnt),
      x, y, z);
}

static void smooth(const std::unique_ptr<CudaSurface<uint8_t>> &active,
                   std::unique_ptr<CudaSurface<float>> &u,
                   std::unique_ptr<CudaSurface<float>> &uBuf,
                   std::unique_ptr<CudaSurface<float>> &b,
                   int n) {
  for (int iter = 0; iter < kSmoothingIters; iter++) {
    DampedJacobiKernel<<<LAUNCH_THREADS_3D(n, n, n)>>>(u->surfaceAccessor(), uBuf->surfaceAccessor(),
                                                       active->surfaceAccessor(), b->surfaceAccessor(), n);
    std::swap(u, uBuf);
  }
}
static __global__ void BottomSolveKernel(CudaSurfaceAccessor<float> u,
                                         CudaSurfaceAccessor<float> b,
                                         CudaSurfaceAccessor<uint8_t> active,
                                         uint n) {
  int tid = ktid(x);
  int x = tid / n;
  int y = (tid - x * n) / n;
  int z = tid % n;
  __shared__ float u_shared[2][8][8][8];
  __shared__ float b_shared[8][8][8];
  __shared__ uint8_t active_shared[8][8][8];
  uint8_t cur = 0;
  u_shared[cur][x][y][z] = u.read(x, y, z);
  b_shared[x][y][z] = b.read(x, y, z);
  active_shared[x][y][z] = active.read(x, y, z);
  __syncthreads();
  for (int i = 0; i < kBottomSolveIters; i++) {
    float u_old = u.read(x, y, z);
    uint8_t axp = active_shared[max(x - 1, 0)][y][z];
    uint8_t axn = active_shared[min(x + 1, n - 1)][y][z];
    uint8_t ayp = active_shared[x][max(y - 1, 0)][z];
    uint8_t ayn = active_shared[x][min(y + 1, n - 1)][z];
    uint8_t azp = active_shared[x][y][max(z - 1, 0)];
    uint8_t azn = active_shared[x][y][min(z + 1, n - 1)];
    auto cnt = static_cast<double>(axp + axn + ayp + ayn + azp + azn);
    float pxp = static_cast<float>(axp) * u_shared[cur][max(x - 1, 0)][y][z];
    float pxn = static_cast<float>(axn) * u_shared[cur][min(x + 1, n - 1)][y][z];
    float pyp = static_cast<float>(ayp) * u_shared[cur][x][max(y - 1, 0)][z];
    float pyn = static_cast<float>(ayn) * u_shared[cur][x][min(y + 1, n - 1)][z];
    float pzp = static_cast<float>(azp) * u_shared[cur][x][y][max(z - 1, 0)];
    float pzn = static_cast<float>(azn) * u_shared[cur][x][y][min(z + 1, n - 1)];
    float div = b_shared[x][y][z];
    u_shared[cur ^ 1][x][y][z] = (1.0 - kDampedJacobiOmega) * static_cast<double>(u_old) +
        kDampedJacobiOmega * static_cast<double>((pxp + pxn + pyp + pyn + pzp + pzn - div) / cnt);
    cur ^= 1;
    __syncthreads();
  }
  u.write(u_shared[cur][x][y][z], x, y, z);
}
// assume: n is the power of 2
// then for a bottom solve which is small enough, we can fit all the data into the shared memory
// and solve them using iterations in one kernel with a warp
static void bottomSolve(const std::unique_ptr<CudaSurface<uint8_t>> &active,
                        const std::unique_ptr<CudaSurface<float>> &u,
                        const std::unique_ptr<CudaSurface<float>> &b,
                        int n) {
  if (n > 8)
    ERROR("bottom solve with n > 8 is not supported yet");
  BottomSolveKernel<<<1, n * n * n>>>(u->surfaceAccessor(), b->surfaceAccessor(),
                                      active->surfaceAccessor(), n);
}
void vCycle(std::array<std::unique_ptr<CudaSurface<uint8_t >>, kVcycleLevel> &active,
            std::array<std::unique_ptr<CudaSurface<float >>, kVcycleLevel> &u,
            std::array<std::unique_ptr<CudaSurface<float >>, kVcycleLevel> &uBuf,
            std::array<std::unique_ptr<CudaSurface<float >>, kVcycleLevel> &b,
            int n) {
  for (int l = 0; l < kVcycleLevel; l++) {
    int N = n >> l;
    smooth(active[l], u[l], uBuf[l], b[l], N);
    ComputeResidualKernel<<<LAUNCH_THREADS_3D(N, N, N)>>>(u[l]->surfaceAccessor(), b[l]->surfaceAccessor(),
                                                          uBuf[l]->surfaceAccessor(), active[l]->surfaceAccessor(), n);
    RestrictKernel<<<LAUNCH_THREADS_3D(N >> 1, N >> 1, N >> 1)>>>(uBuf[l]->surfaceAccessor(),
                                                                  uBuf[l + 1]->surfaceAccessor(),
                                                                  n);
  }
  bottomSolve(active[kVcycleLevel], u[kVcycleLevel], b[kVcycleLevel], n >> kVcycleLevel);
  for (int l = kVcycleLevel - 1; l >= 0; l--) {
    int N = n >> l;
    ProlongateKernel<<<LAUNCH_THREADS_3D(N, N, N)>>>(u[l + 1]->surfaceAccessor(),
                                                     active[l]->surfaceAccessor(),
                                                     u[l]->surfaceAccessor(), n);
    smooth(active[l], u[l], uBuf[l], b[l], N);
  }
}

void prepareWeights() {
  double weights[4][4][4];
  for (auto &wi : weights)
    for (auto &wij : wi)
      for (double &wijk : wij)
        wijk = 1.0;
  for (int i = 0; i < 4; i++)
    for (auto &wij : weights[i])
      for (auto &wijk : wij) {
        if (i == 0 || i == 3) wijk *= 0.125;
        else if (i == 1 || i == 2) wijk *= 0.375;
      }
  for (auto &wi : weights)
    for (int j = 0; j < 4; j++)
      for (auto &wijk : wi[j]) {
        if (j == 0 || j == 3) wijk *= 0.125;
        else if (j == 1 || j == 2) wijk *= 0.375;
      }
  for (auto &wi : weights)
    for (auto &wij : wi)
      for (int k = 0; k < 4; k++) {
        if (k == 0 || k == 3) wij[k] *= 0.125;
        else if (k == 1 || k == 2) wij[k] *= 0.375;
      }
  hipMemcpyToSymbol(HIP_SYMBOL(kTransferWights), weights, sizeof(weights));
}
}